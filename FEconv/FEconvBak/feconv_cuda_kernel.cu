#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

template <typename scalar_t>
__global__ void feconv_cuda_forward_kernel2(
	const torch::PackedTensorAccessor<scalar_t,5,torch::RestrictPtrTraits,size_t> U,
	const torch::PackedTensorAccessor<int,5,torch::RestrictPtrTraits,size_t> H8types,
	const torch::PackedTensorAccessor<int,5,torch::RestrictPtrTraits,size_t> nodIdx,
	const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> filters,
	torch::PackedTensorAccessor<scalar_t,5,torch::RestrictPtrTraits,size_t> KU)
{	
	const int outidx = blockIdx.x / 41; // 0 - 17
	const int Idxx = blockIdx.x % 41; // 0 - 40
	//const int Idxx = threadIdx.x % 41;
	//const int Idxy = threadIdx.x / 41;
	// const int Idxx = blockIdx.x;
	const int Idxy = blockIdx.y;
	const int Idxz = blockIdx.z;

	const int h8type = H8types[threadIdx.x][0][Idxx][Idxy][Idxz];
	//const auto fkernels = filters[h8type];

	scalar_t convresult = 0.0;

	int direction = outidx % 3;

	for (int ix= 0; ix < 3; ix++)
	{
		for (int j = 0; j < 27; j++)
		{
			int uidx1 = nodIdx[Idxx][Idxy][Idxz][j][0];
			int uidx2 = nodIdx[Idxx][Idxy][Idxz][j][1];
			int uidx3 = nodIdx[Idxx][Idxy][Idxz][j][2];
			if ((uidx1+1)*(uidx2+1)*(uidx3+1)!=0)
			{
				convresult += U[threadIdx.x][outidx - direction + ix][uidx1][uidx2][uidx3] * filters[h8type][3 * direction + ix][j];
			}
		}
	}
	KU[threadIdx.x][outidx][Idxx][Idxy][Idxz] = convresult;
}


template <typename scalar_t>
__global__ void feconv_cuda_forward_kernel(
	const torch::PackedTensorAccessor<scalar_t,5,torch::RestrictPtrTraits,size_t> U,
	const torch::PackedTensorAccessor<int,5,torch::RestrictPtrTraits,size_t> H8types,
	const torch::PackedTensorAccessor<int,5,torch::RestrictPtrTraits,size_t> nodIdx,
	const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> filters,
	torch::PackedTensorAccessor<scalar_t,5,torch::RestrictPtrTraits,size_t> KU)
{	
	//const int Idxx = threadIdx.x % 41;
	//const int Idxy = threadIdx.x / 41;
	const int Idxx = blockIdx.x;
	const int Idxy = blockIdx.y;
	const int Idxz = blockIdx.z;
	const int h8type = H8types[threadIdx.y][0][Idxx][Idxy][Idxz];
	const auto fkernels = filters[h8type];

	scalar_t convresult = 0.0;

	int direction = threadIdx.x % 3;
/**/
	for (int ix= 0; ix < 3; ix++)
	{
		for (int j = 0; j < 27; j++)
		{
			int uidx1 = nodIdx[Idxx][Idxy][Idxz][j][0];
			int uidx2 = nodIdx[Idxx][Idxy][Idxz][j][1];
			int uidx3 = nodIdx[Idxx][Idxy][Idxz][j][2];
			if ((uidx1+1)*(uidx2+1)*(uidx3+1)!=0)
			{
				convresult += U[threadIdx.y][threadIdx.x - direction + ix][uidx1][uidx2][uidx3] * filters[h8type][3 * direction + ix][j];
//				convresult += U[threadIdx.y][threadIdx.x - direction + ix][uidx1][uidx2][uidx3] * filters[255][3 * direction + ix][j];
//				convresult += filters[h8type][3 * direction + ix][j];
			}
		}
	}
	//KU[blockIdx.x][blockIdx.y][threadIdx.x][threadIdx.y][threadIdx.z] = 1.0;
	KU[threadIdx.y][threadIdx.x][Idxx][Idxy][Idxz] = convresult;
	/*
	const int h8type = H8types[blockIdx.y][blockIdx.x][threadIdx.x][threadIdx.y][threadIdx.z];
	const auto fkernels = filters[h8type];

	scalar_t convresult = 0.0;

	//blockIdx.y % 3 == 0: filters[0:3]
	//blockIdx.y % 3 == 1: filters[3:6]
	//blockIdx.y % 3 == 2: filters[6:9]
	int direction = blockIdx.x % 3;

	for (int ix= 0; ix < 3; ix++)
	{
		for (int j = 0; j < 27; j++)
		{
			int uidx1 = nodIdx[threadIdx.x][threadIdx.y][threadIdx.z][j][0];
			int uidx2 = nodIdx[threadIdx.x][threadIdx.y][threadIdx.z][j][1];
			int uidx3 = nodIdx[threadIdx.x][threadIdx.y][threadIdx.z][j][2];
			convresult += 1.0;//U[blockIdx.x][blockIdx.y - direction + ix][uidx1][uidx2][uidx3] * filters[h8type][3 * direction + ix][j];
		}
	}
	// scalar_t convresult = h8type*1.0;
	//KU[blockIdx.x][blockIdx.y][threadIdx.x][threadIdx.y][threadIdx.z] = 1.0;
	KU[blockIdx.y][blockIdx.x][threadIdx.x][threadIdx.y][threadIdx.z] = convresult;*/
}



std::vector<torch::Tensor> 
//torch::Tensor 
feconv_cuda_forward(
	torch::Tensor U,
	torch::Tensor H8types,
	torch::Tensor nodIdx,
	torch::Tensor filters)
{

	const auto batch_size = U.size(0);

	auto KU = torch::zeros_like(U);


	// const dim3 blocks(41,41,41);
	// const dim3 threads(18,batch_size);
	const dim3 blocks(18*41,41,41);
	const dim3 threads(batch_size);

	//const dim3 blocks(18,batch_size);
	//const dim3 threads(41,41,41);
	// const dim3 blocks(batch_size,3);
	// const dim3 threads(11,11,11);
	/**/
	AT_DISPATCH_FLOATING_TYPES(U.type(), "feconv_forward_cuda", ([&] {
    feconv_cuda_forward_kernel2<scalar_t><<<blocks, threads>>>(
        U.packed_accessor<scalar_t,5,torch::RestrictPtrTraits,size_t>(),
        H8types.packed_accessor<int,5,torch::RestrictPtrTraits,size_t>(),
        nodIdx.packed_accessor<int,5,torch::RestrictPtrTraits,size_t>(),
        filters.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
        KU.packed_accessor<scalar_t,5,torch::RestrictPtrTraits,size_t>());
  }));

  return {KU};
  //return {U};
}

std::vector<torch::Tensor> 
// torch::Tensor 
feconv_cuda_backward(
	torch::Tensor gradU)
{
	auto dU = torch::zeros_like(gradU);
	return {dU};
}
